#include "hip/hip_runtime.h"
#include "deviceCode.h"
#include "dda.h"
#include <optix_device.h>
#include "renderer.h"
#include "unstructuredElementHelper.h"

using namespace owl;
using namespace dtracker;

extern "C" __constant__ LaunchParams optixLaunchParams;

#define DEBUG 1
// create a debug function macro that gets called only for center pixel
inline __device__ bool dbg()
{
#if DEBUG
    return false;
#else
    auto lp = optixLaunchParams;
    auto pixelID = vec2i(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
    return (lp.fbSize.x / 2 == pixelID.x) &&
           (lp.fbSize.y / 2 == pixelID.y);
#define ACTIVATE_CROSSHAIRS
#endif
}

inline __both__ float4 transferFunction(float f, size_t tfID = 0)
{
    auto &lp = optixLaunchParams;
    if (f < lp.transferFunction[tfID].volumeDomain.x ||
        f > lp.transferFunction[tfID].volumeDomain.y)
    {
        return make_float4(1.f, 0.f, 1.f, 0.0f);
    }
    float remapped1 = (f - lp.transferFunction[tfID].volumeDomain.x) / (lp.transferFunction[tfID].volumeDomain.y - lp.transferFunction[tfID].volumeDomain.x);
    float remapped2 = (remapped1 - lp.transferFunction[tfID].xfDomain.x) / (lp.transferFunction[tfID].xfDomain.y - lp.transferFunction[tfID].xfDomain.x);
    
    float4 xf = tex2D<float4>(lp.transferFunction[tfID].xf, remapped2, 0.5f);
    xf.w *= lp.transferFunction[tfID].opacityScale;

    return xf;
}

inline __device__ vec3f over(vec3f Cin, vec3f Cx, float Ain, float Ax)
{
  return Cin + Cx * Ax * (1.f - Ain);
}

inline __device__ float over(const float Ain, const float Ax)
{
  return Ain + (1.f - Ain) * Ax;
}

inline __device__ vec4f over(const vec4f &in, const vec4f &x)
{
  auto c = over(vec3f(in), vec3f(x), in.w, x.w);
  auto a = over(in.w, x.w);
  return vec4f(c, a);
}

inline __device__ void generateRay(const vec2f screen, owl::Ray &ray)
{
    auto &lp = optixLaunchParams;
    ray.origin = lp.camera.origin;
    vec3f direction = lp.camera.lower_left_corner +
                      screen.u * lp.camera.horizontal +
                      screen.v * lp.camera.vertical;
    // direction = normalize(direction);
    if (fabs(direction.x) < 1e-5f)
        direction.x = 1e-5f;
    if (fabs(direction.y) < 1e-5f)
        direction.y = 1e-5f;
    if (fabs(direction.z) < 1e-5f)
        direction.z = 1e-5f;
    ray.direction = normalize(direction - ray.origin);
}

inline __device__
vec3f missCheckerBoard(const vec3f& color0 = vec3f(.2f, .2f, .26f), 
    const vec3f& color1 = vec3f(.1f, .1f, .16f), int gap = 25)
{
    const vec2i pixelID = owl::getLaunchIndex();

    int pattern = (pixelID.x / gap) ^ (pixelID.y / gap);
    vec3f color = (pattern & 1) ? color1 : color0;
    return color;
}

inline __device__
float sampleVolumeTexture(const vec3f& normalizedPos, const int meshID = 0)
{
    auto &lp = optixLaunchParams;
    float value = tex3D<float>(lp.volume.sGrid[meshID].scalarTex, 
            normalizedPos.x, normalizedPos.y,normalizedPos.z);
                    
    // Sample scalar field
    return value;
}

inline __device__
float sampleVolume(const vec3f& pos, const int meshID = 0)
{
    auto &lp = optixLaunchParams;
    if(lp.volume.meshType == 1)//Query unstructred mesh
    {
        //create a ray with zero lenght and origin at pos
        Ray ray;
        ray.origin = pos;
        ray.direction = vec3f(1.0f, 1.0f, 1.0f);
        ray.tmin = 0.0f;
        ray.tmax = 0.0f;
        ray.time = 0.0f;
        RayPayload prd;
        prd.debug = dbg();

        owl::traceRay(lp.volume.elementTLAS, ray, prd);
        if (prd.missed)
            return NAN;
        else
            return prd.dataValue;
    }
    else if(lp.volume.meshType == 2)//Query structured mesh
    {
        //normalize pos to [0,1] using bounds of voxel grid
        vec3f normalizedPos = (pos - vec3f(lp.volume.globalBoundsLo)) / 
            (vec3f(lp.volume.globalBoundsHi) - vec3f(lp.volume.globalBoundsLo)); 
        // Sample scalar field
        return sampleVolumeTexture(normalizedPos, meshID);
    }
}


OPTIX_RAYGEN_PROGRAM(mainRG)
()
{
    auto &lp = optixLaunchParams;
    const vec2i pixelID = owl::getLaunchIndex();
    const int fbOfs = pixelID.x + lp.fbSize.x * pixelID.y;

    //generate ray
    int seed = owl::getLaunchDims().x * owl::getLaunchDims().y * lp.frameID;
    owl::common::LCG<4> random(threadIdx.x + seed, threadIdx.y + seed);//jittered sampling
    const vec2f screen = (vec2f(pixelID) + random()) / vec2f(lp.fbSize);
    Ray ray;
    generateRay(screen, ray);

    //test surface intersections first
    RayPayload surfPrd;
    vec4f finalColor = vec4f(lp.bgColor,1.f);//vec4f(missCheckerBoard(), 1.0f);
    vec4f color = vec4f(0.0f, 0.0f, 0.0f, 0.0f);
    traceRay(lp.triangleTLAS, ray, surfPrd, OPTIX_RAY_FLAG_DISABLE_ANYHIT); //surface
    if (!surfPrd.missed)
         color = surfPrd.rgba;

    const float tMax = surfPrd.missed ? 1e20 : surfPrd.tHit;

    //test for root macrocell intersection
    RayPayload volumePrd;
    volumePrd.debug = dbg();
    volumePrd.t0 = 0.f;
    volumePrd.t1 = tMax;
    traceRay(lp.volume.rootMacrocellTLAS, ray, volumePrd, OPTIX_RAY_FLAG_DISABLE_ANYHIT); //root macrocell to initiate dda traversal
    if(!volumePrd.missed)
    {
        vec3f albedo = vec3f(volumePrd.rgba);
        if(lp.enableShadows)
        {
            // trace shadow rays
            RayPayload shadowbyVolPrd;
            shadowbyVolPrd.debug = dbg();
            shadowbyVolPrd.t0 = 0.f;
            shadowbyVolPrd.t1 = 1e20f; //todo fix this

            Ray shadowRay;
            shadowRay.origin = ray.origin + volumePrd.tHit * ray.direction;
            shadowRay.direction = -lp.lightDir;
            shadowRay.tmin = 0.00f;
            shadowRay.tmax = 1e20f;

            traceRay(lp.volume.rootMacrocellTLAS, shadowRay, shadowbyVolPrd, OPTIX_RAY_FLAG_DISABLE_ANYHIT);
            vec3f shadow((1.f - lp.ambient) * (1.f - shadowbyVolPrd.rgba.w)  + lp.ambient);
            color = vec4f(albedo * shadow * lp.lightIntensity, 1.0f);

            volumePrd.samples += shadowbyVolPrd.samples;// for heatmap
            volumePrd.rejections += shadowbyVolPrd.rejections;// for heatmap
        }
        else
            color = vec4f(albedo * lp.lightIntensity, 1.0f);
    }

    if(lp.heatMapMode == 1)
    {
        //heatmap
        int samples = volumePrd.samples * (lp.volume.meshType == 0 ? 1 : 10 / lp.volume.numMeshes);
        lp.fbPtr[fbOfs] = make_rgba(vec4f(samples / 250.f, samples / 250.f, samples / 250.f, 1.f));
    }
    else if(lp.heatMapMode == 2)
    {
        //heatmap
        int rejections = volumePrd.rejections * (lp.volume.meshType == 0 ? 1 : 10 / lp.volume.numMeshes);
        lp.fbPtr[fbOfs] = make_rgba(vec4f(rejections / 250.f, rejections / 250.f, rejections / 250.f, 1.f));
    }
    else
    {
        finalColor = over(color, finalColor);
        if(lp.enableAccumulation)
        {
            const vec4f accumColor = lp.accumBuffer[fbOfs];
            finalColor = (vec4f(finalColor) + float(lp.accumID) * accumColor) / float(lp.accumID + 1);
            lp.fbPtr[fbOfs] = make_rgba(vec4f(finalColor));
            lp.accumBuffer[fbOfs] = vec4f(finalColor);
        }
        else
            lp.fbPtr[fbOfs] = make_rgba(vec4f(finalColor));
#ifdef ACTIVATE_CROSSHAIRS
    if (pixelID.x == lp.fbSize.x / 2 || pixelID.y == lp.fbSize.y / 2 || 
        pixelID.x == lp.fbSize.x / 2 + 1 || pixelID.y == lp.fbSize.y / 2 + 1 || 
        pixelID.x == lp.fbSize.x / 2 - 1 || pixelID.y == lp.fbSize.y / 2 - 1)
        lp.fbPtr[fbOfs] = make_rgba(vec4f(1.0f-finalColor.x, 1.0f-finalColor.y, 1.0f-finalColor.y, 1.f));
#endif
    }
}

OPTIX_CLOSEST_HIT_PROGRAM(triangleCH)
()
{
    // get light direction and do a simple lambert shading

    const TriangleData &self = owl::getProgramData<TriangleData>();

    // compute normal:
    const int primID = optixGetPrimitiveIndex();
    const vec3i index = self.indices[primID];
    const vec3f &A = self.vertices[index.x];
    const vec3f &B = self.vertices[index.y];
    const vec3f &C = self.vertices[index.z];
    const vec3f Ng = normalize(cross(B - A, C - A));
    const vec2f bary = optixGetTriangleBarycentrics();
    const vec3f P = bary.x * A + bary.y * B + (1.f - bary.x - bary.y) * C;

    const vec3f rayDir = optixGetWorldRayDirection();
    RayPayload &prd = owl::getPRD<RayPayload>();
    prd.tHit = length(P - vec3f(optixGetWorldRayOrigin()));
    prd.missed = false;
    prd.rgba = vec4f((.2f + .8f * fabs(dot(rayDir, Ng))) * self.color, 1);
}


//define an enum for different volume rendering events
// enum VolumeEvent
// {
//     ABSORPTION,
//     SCATTERING,
//     NULL_COLLISION // also used as no collision
// };
OPTIX_CLOSEST_HIT_PROGRAM(adaptiveDTCH)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    auto &lp = optixLaunchParams;
    const MacrocellData &self = owl::getProgramData<MacrocellData>();
    prd.missed = true;
    prd.rgba = vec4f(0.0f, 0.0f, 0.0f, 0.0f);

    float unit = lp.volume.globalOpacity;

    vec3f worldOrg = optixGetWorldRayOrigin();
    vec3f org = optixGetWorldRayOrigin();
    vec3f worldDir = optixGetWorldRayDirection();

    // assuming ray is already in voxel space
    box3f worlddim = {{lp.volume.globalBoundsLo.x, lp.volume.globalBoundsLo.y, lp.volume.globalBoundsLo.z},
                      {lp.volume.globalBoundsHi.x, lp.volume.globalBoundsHi.y, lp.volume.globalBoundsHi.z}};
    vec3ui mcDim = lp.volume.macrocellDims;

    org = org - worlddim.lower;

    const vec3f worldToUnit = 1.f / (worlddim.upper - worlddim.lower);
    const vec3f unitToGrid = vec3f(mcDim.x, mcDim.y, mcDim.z);
    
    org = unitToGrid *worldToUnit * org;
    vec3f dir = worldToUnit * unitToGrid * worldDir;
    const float gridToWorldT = 1.f / length(dir);
    dir = normalize(dir);

    //VolumeEvent event = NULL_COLLISION;
    auto lambda = [&](const vec3i &cellIdx, float t0, float t1) -> bool
    {
        const int cellID = cellIdx.x + cellIdx.y * mcDim.x + cellIdx.z * mcDim.x * mcDim.y;
        float majorant = lp.volume.majorants[cellID];

        if(prd.debug)
            printf("cellID = %d, majorant = %f\n", cellID, majorant);

        if (majorant == 0.00f)
            return true;

        float t = t0;

        // Sample free-flight distance
        while (true)
        {
            //t_{i} = t_{i-1} - ln(1-rand())/mu_{t,max}
            //NOTE: this "unit" can be considered as a global opacity scale ass it makes sampling a point
            // more/less probable by altering the length of the woodcock step size
            t = t - (log(1.0f - prd.rng()) / majorant) * unit;

            // A cell boundary has been hit
            if (t >= t1){
                //event = NULL_COLLISION;
                break; // go to next cell
            }

            // Update current position
            const float tWorld = t * gridToWorldT;
            const vec3f xTexture = (worldOrg + tWorld * worldDir) * worldToUnit;
            // A world boundary has been hit
            if (tWorld >= prd.t1)
            {
                //event = NULL_COLLISION;
                prd.rejections++;
                return false; // terminate traversal
            }
            
            //density(w component of float4) at TF(ray(t)) similar to spectrum(TR * 1 - max(0, density * invMaxDensity)) in pbrt
            //get values from all meshes and decide which one the sample is gonna come from
            float meshSelector = prd.rng() * majorant;
            for(int meshID = 0; meshID < lp.volume.numMeshes; meshID++)
            {
                const float value = sampleVolumeTexture(xTexture, meshID);
                prd.samples++;
                if(isnan(value)) // miss: this shouldnt happen in structured volumes
                {
                    //event = NULL_COLLISION;
                    continue;
                }
                const float4 curSample = transferFunction(value, meshID);
                //sample a mesh based on its opacity
                if(curSample.w > 0.0f && meshSelector < curSample.w)
                {
                    //event = ABSORPTION;
                    prd.tHit = tWorld;
                    prd.rgba = curSample;
                    prd.rgba.w = 1.0f;
                    prd.missed = false;
                    return false;
                }
                meshSelector -= curSample.w;
            }
            //if the process survies all meshes, it is a null collision, keep going
            //event = NULL_COLLISION;
            prd.rejections++;
        }

        return true;
    };
    dda::dda3(org,dir,1e20f,mcDim,lambda,false);
}

OPTIX_CLOSEST_HIT_PROGRAM(adaptiveMMDTCH)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    auto &lp = optixLaunchParams;
    const MacrocellData &self = owl::getProgramData<MacrocellData>();
    prd.missed = true;
    prd.rgba = vec4f(0.0f, 0.0f, 0.0f, 0.0f);

    float unit = lp.volume.globalOpacity;

    vec3f worldOrg = optixGetWorldRayOrigin();
    vec3f org = optixGetWorldRayOrigin();
    vec3f worldDir = optixGetWorldRayDirection();

    // assuming ray is already in voxel space
    box3f worlddim = {{lp.volume.globalBoundsLo.x, lp.volume.globalBoundsLo.y, lp.volume.globalBoundsLo.z},
                      {lp.volume.globalBoundsHi.x, lp.volume.globalBoundsHi.y, lp.volume.globalBoundsHi.z}};
    vec3ui mcDim = lp.volume.macrocellDims;

    org = org - worlddim.lower;

    const vec3f worldToUnit = 1.f / (worlddim.upper - worlddim.lower);
    const vec3f unitToGrid = vec3f(mcDim.x, mcDim.y, mcDim.z);
    
    org = unitToGrid *worldToUnit * org;
    vec3f dir = worldToUnit * unitToGrid * worldDir;
    const float gridToWorldT = 1.f / length(dir);
    dir = normalize(dir);

    float majorants[MAX_MESHES];
    float ts[MAX_MESHES];
    //VolumeEvent event = NULL_COLLISION;
    auto lambda = [&](const vec3i &cellIdx, float t0, float t1) -> bool
    {
        const int cellID = cellIdx.x + cellIdx.y * mcDim.x + cellIdx.z * mcDim.x * mcDim.y;

        float majorantSum = 0.0f;

        for (int i = 0; i < lp.volume.numMeshes; i++)
        {
            majorants[i] = lp.volume.majorants[cellID * lp.volume.numMeshes + i];
            majorantSum += majorants[i];
            ts[i] = t0;
        }

        if(prd.debug)
            for (int i = 0; i < lp.volume.numMeshes; i++)
                printf("cellID = %d, majorant = %f\n", cellID, majorants[i]);

        if (majorantSum == 0.00f)
            return true;

        for (int i = 0; i < lp.volume.numMeshes; i++)
            ts[i] = ts[i] - (log(1.0f - prd.rng()) / majorants[i]) * unit;

        // Sample free-flight distance
        while (true)
        {
            //t_{i} = t_{i-1} - ln(1-rand())/mu_{t,max}
            //NOTE: this "unit" can be considered as a global opacity scale ass it makes sampling a point
            // more/less probable by altering the length of the woodcock step size
            float minT = ts[0];
            int selectedChannel = 0;
            auto rand = prd.rng();
            for (int i = 1; i < lp.volume.numMeshes; i++)
            {
                if(ts[i] < minT)
                {
                    selectedChannel = i;
                    minT = ts[i];
                }
            }

            // A cell boundary has been hit
            if (minT >= t1){
                //event = NULL_COLLISION;
                break; // go to next cell
            }

            // Update current position
            const float tWorld = minT * gridToWorldT;
            const vec3f xTexture = (worldOrg + tWorld * worldDir) * worldToUnit;
            // A world boundary has been hit
            if (tWorld >= prd.t1)
            {
                //event = NULL_COLLISION;
                prd.rejections++;
                return false; // terminate traversal
            }
            
            //density(w component of float4) at TF(ray(t)) similar to spectrum(TR * 1 - max(0, density * invMaxDensity)) in pbrt
            //get values from all meshes and decide which one the sample is gonna come from
            float meshSelector = prd.rng() * (majorants[selectedChannel]);
            const float value = sampleVolumeTexture(xTexture, selectedChannel);
            prd.samples++;
            if(isnan(value)) // miss: this shouldnt happen in structured volumes
            {
                //event = NULL_COLLISION;
                continue;
            }
            const float4 curSample = transferFunction(value, selectedChannel);
            //sample a mesh based on its opacity
            if(curSample.w > 0.0f && meshSelector < curSample.w)
            {
                //event = ABSORPTION;
                prd.tHit = tWorld;
                prd.rgba = curSample;
                prd.rgba.w = 1.0f;
                prd.missed = false;
                return false;
            }
            //if the process survies all meshes, it is a null collision, keep going
            //event = NULL_COLLISION;
            prd.rejections++;
            ts[selectedChannel] = ts[selectedChannel] - (log(1.0f - rand) / majorants[selectedChannel]) * unit;
        }

        return true;
    };
    dda::dda3(org,dir,1e20f,mcDim,lambda,false);
}

OPTIX_CLOSEST_HIT_PROGRAM(adaptiveBaseLineDTCH)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    auto &lp = optixLaunchParams;
    const MacrocellData &self = owl::getProgramData<MacrocellData>();
    prd.missed = true;
    prd.rgba = vec4f(0.0f, 0.0f, 0.0f, 0.0f);

    float unit = lp.volume.globalOpacity;

    vec3f worldOrg = optixGetWorldRayOrigin();
    vec3f org = optixGetWorldRayOrigin();
    vec3f worldDir = optixGetWorldRayDirection();

    // assuming ray is already in voxel space
    box3f worlddim = {{lp.volume.globalBoundsLo.x, lp.volume.globalBoundsLo.y, lp.volume.globalBoundsLo.z},
                      {lp.volume.globalBoundsHi.x, lp.volume.globalBoundsHi.y, lp.volume.globalBoundsHi.z}};
    vec3ui mcDim = lp.volume.macrocellDims;

    org = org - worlddim.lower;

    const vec3f worldToUnit = 1.f / (worlddim.upper - worlddim.lower);
    const vec3f unitToGrid = vec3f(mcDim.x, mcDim.y, mcDim.z);
    
    org = unitToGrid *worldToUnit * org;
    vec3f dir = worldToUnit * unitToGrid * worldDir;
    const float gridToWorldT = 1.f / length(dir);
    //const float worldToUnitT = owl::length(mcDim) / length(worlddim.upper - worlddim.lower);
    dir = normalize(dir);

    float majorants[MAX_MESHES];
    float ts[MAX_MESHES];
    int curMesh = 0;
    float tMax = 1e20f;
    
    //VolumeEvent event = NULL_COLLISION;
    auto lambda = [&](const vec3i &cellIdx, float t0, float t1) -> bool
    {
        const int cellID = cellIdx.x + cellIdx.y * mcDim.x + cellIdx.z * mcDim.x * mcDim.y;
        float majorant = lp.volume.majorants[cellID * lp.volume.numMeshes + curMesh];

        if(prd.debug)
            printf("cellID = %d, majorant = %f\n", cellID, majorant);

        if (majorant == 0.00f)
            return true;

        float t = t0;

        // Sample free-flight distance
        while (true)
        {
            //t_{i} = t_{i-1} - ln(1-rand())/mu_{t,max}
            //NOTE: this "unit" can be considered as a global opacity scale ass it makes sampling a point
            // more/less probable by altering the length of the woodcock step size
            t = t - (log(1.0f - prd.rng()) / majorant) * unit;

            // A cell boundary has been hit
            if (t >= t1){
                //event = NULL_COLLISION;
                break; // go to next cell
            }

            if(tMax < t)
                return false;

            // Update current position
            const float tWorld = t * gridToWorldT;
            const vec3f xTexture = (worldOrg + tWorld * worldDir) * worldToUnit;
            // A world boundary has been hit
            if (tWorld >= prd.t1)
            {
                //event = NULL_COLLISION;
                prd.rejections++;
                return false; // terminate traversal
            }
            
            //density(w component of float4) at TF(ray(t)) similar to spectrum(TR * 1 - max(0, density * invMaxDensity)) in pbrt
            //get values from all meshes and decide which one the sample is gonna come from
            float meshSelector = prd.rng() * majorant;
            const float value = sampleVolumeTexture(xTexture, curMesh);
            prd.samples++;
            if(isnan(value)) // miss: this shouldnt happen in structured volumes
            {
                //event = NULL_COLLISION;
                continue;
            }
            const float4 curSample = transferFunction(value, curMesh);
            //sample a mesh based on its opacity
            if(curSample.w > 0.0f && meshSelector < curSample.w)
            {
                //event = ABSORPTION;
                prd.tHit = tWorld;
                prd.rgba = curSample;
                prd.rgba.w = 1.0f;
                prd.missed = false;
                tMax = min(tMax, t);
                return false;
            }
            //if the process survies all meshes, it is a null collision, keep going
            //event = NULL_COLLISION;
            prd.rejections++;
        }

        return true;
    };
    for (int i = 0; i < lp.volume.numMeshes; i++)
    {
        curMesh = i;
        dda::dda3(org,dir,1e20,mcDim,lambda,false);
    }
}

OPTIX_MISS_PROGRAM(miss)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    prd.missed = true;
}

// ------------------------------------------------------------------
// Bounds programs for volume elements
// ------------------------------------------------------------------

OPTIX_BOUNDS_PROGRAM(macrocellBounds)
(
    const void *geomData,
    owl::common::box3f &primBounds,
    const int primID)
{
    const MacrocellData &self = *(const MacrocellData *)geomData;
    // if (self.maxima[primID] <= 0.f) {
    //    primBounds = box3f(); // empty box
    //  }
    //  else
    {
        primBounds = box3f();
        primBounds = primBounds.including(vec3f(self.bboxes[(primID)].lower.x,
                                                self.bboxes[(primID)].lower.y,
                                                self.bboxes[(primID)].lower.z));
        primBounds = primBounds.including(vec3f(self.bboxes[(primID)].upper.x,
                                                self.bboxes[(primID)].upper.y,
                                                self.bboxes[(primID)].upper.z));
        // primBounds.lower.x = self.bboxes[(primID * 2 + 0)].x;
        // primBounds.lower.y = self.bboxes[(primID * 2 + 0)].y;
        // primBounds.lower.z = self.bboxes[(primID * 2 + 0)].z;
        // primBounds.upper.x = self.bboxes[(primID * 2 + 1)].x;
        // primBounds.upper.y = self.bboxes[(primID * 2 + 1)].y;
        // primBounds.upper.z = self.bboxes[(primID * 2 + 1)].z;
    }
}

OPTIX_BOUNDS_PROGRAM(tetrahedraBounds)
(
    const void *geomData,
    owl::common::box3f &primBounds,
    const int primID)
{
    const UnstructuredElementData &self = *(const UnstructuredElementData *)geomData;
    primBounds = box3f();
    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;
    if (ID >= self.numTetrahedra)
        return;

    unsigned int *tets = (unsigned int *)self.tetrahedra;
    uint64_t i0 = tets[ID * 4 + 0];
    uint64_t i1 = tets[ID * 4 + 1];
    uint64_t i2 = tets[ID * 4 + 2];
    uint64_t i3 = tets[ID * 4 + 3];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];

    primBounds = primBounds.including(P0)
                     .including(P1)
                     .including(P2)
                     .including(P3);
}

OPTIX_BOUNDS_PROGRAM(pyramidBounds)
(
    const void *geomData,
    owl::common::box3f &primBounds,
    const int primID)

{
    const UnstructuredElementData &self = *(const UnstructuredElementData *)geomData;
    primBounds = box3f();
    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;
    if (ID >= self.numPyramids)
        return;

    unsigned int *pyrs = (unsigned int *)self.pyramids;
    uint64_t i0 = pyrs[ID * 5 + 0];
    uint64_t i1 = pyrs[ID * 5 + 1];
    uint64_t i2 = pyrs[ID * 5 + 2];
    uint64_t i3 = pyrs[ID * 5 + 3];
    uint64_t i4 = pyrs[ID * 5 + 4];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];
    vec3f P4 = self.vertices[i4];

    primBounds = primBounds.including(P0)
                     .including(P1)
                     .including(P2)
                     .including(P3)
                     .including(P4);
}

OPTIX_BOUNDS_PROGRAM(wedgeBounds)
(
    const void *geomData,
    owl::common::box3f &primBounds,
    const int primID)
{
    const UnstructuredElementData &self = *(const UnstructuredElementData *)geomData;
    primBounds = box3f();
    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;
    if (ID >= self.numWedges)
        return;

    unsigned int *weds = (unsigned int *)self.wedges;
    uint64_t i0 = weds[ID * 6 + 0];
    uint64_t i1 = weds[ID * 6 + 1];
    uint64_t i2 = weds[ID * 6 + 2];
    uint64_t i3 = weds[ID * 6 + 3];
    uint64_t i4 = weds[ID * 6 + 4];
    uint64_t i5 = weds[ID * 6 + 5];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];
    vec3f P4 = self.vertices[i4];
    vec3f P5 = self.vertices[i5];

    primBounds = primBounds.including(P0)
                     .including(P1)
                     .including(P2)
                     .including(P3)
                     .including(P4)
                     .including(P5);
}

OPTIX_BOUNDS_PROGRAM(hexahedraBounds)
(
    const void *geomData,
    owl::common::box3f &primBounds,
    const int primID)
{
    const UnstructuredElementData &self = *(const UnstructuredElementData *)geomData;
    primBounds = box3f();
    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;
    if (ID >= self.numHexahedra)
        return;

    unsigned int *hexes = (unsigned int *)self.hexahedra;
    uint64_t i0 = hexes[ID * 8 + 0];
    uint64_t i1 = hexes[ID * 8 + 1];
    uint64_t i2 = hexes[ID * 8 + 2];
    uint64_t i3 = hexes[ID * 8 + 3];
    uint64_t i4 = hexes[ID * 8 + 4];
    uint64_t i5 = hexes[ID * 8 + 5];
    uint64_t i6 = hexes[ID * 8 + 6];
    uint64_t i7 = hexes[ID * 8 + 7];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];
    vec3f P4 = self.vertices[i4];
    vec3f P5 = self.vertices[i5];
    vec3f P6 = self.vertices[i6];
    vec3f P7 = self.vertices[i7];
    primBounds.extend(P0)
        .extend(P1)
        .extend(P2)
        .extend(P3)
        .extend(P4)
        .extend(P5)
        .extend(P6)
        .extend(P7);
    // primBounds.extend(P7); // wtf??!
}

// ------------------------------------------------------------------
// intersection programs
// ------------------------------------------------------------------
OPTIX_INTERSECT_PROGRAM(tetrahedraPointQuery)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    const auto &self = owl::getProgramData<UnstructuredElementData>();
    unsigned int primID = optixGetPrimitiveIndex(); //+ self.offset;
    float3 origin = optixGetObjectRayOrigin();

    // for (int i = 0; i < ELEMENTS_PER_BOX; ++i) {
    //   uint32_t ID = primID * ELEMENTS_PER_BOX + i;
    if (primID >= self.numTetrahedra)
        return;

    // printf("TetrahedraPointQuery: primID = %d\\n", primID);

    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;

    vec3f P = {origin.x, origin.y, origin.z};

    // unsigned int i0, i1, i2, i3;
    uint32_t *tets = (uint32_t *)self.tetrahedra;
    uint64_t i0 = tets[ID * 4 + 0];
    uint64_t i1 = tets[ID * 4 + 1];
    uint64_t i2 = tets[ID * 4 + 2];
    uint64_t i3 = tets[ID * 4 + 3];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];

    float S0 = self.scalars[i0];
    float S1 = self.scalars[i1];
    float S2 = self.scalars[i2];
    float S3 = self.scalars[i3];

    // prd.missed = false;              // for
    // prd.dataValue = S0;              // testing
    // optixReportIntersection(0.f, 0); // please
    // return;                          // remove

    if (interpolateTetrahedra(P, P0, P1, P2, P3, S0, S1, S2, S3, prd.dataValue))
    {
        optixReportIntersection(0.f, 0);
        prd.missed = false;
        return;
    }
}

OPTIX_INTERSECT_PROGRAM(pyramidPointQuery)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    const auto &self = owl::getProgramData<UnstructuredElementData>();
    unsigned int primID = optixGetPrimitiveIndex(); //+ self.offset;
    float3 origin = optixGetObjectRayOrigin();

    // for (int i = 0; i < ELEMENTS_PER_BOX; ++i) {
    //   uint32_t ID = primID * ELEMENTS_PER_BOX + i;
    if (primID >= self.numPyramids)
        return;

    // printf("TetrahedraPointQuery: primID = %d\\n", primID);

    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;

    vec3f P = {origin.x, origin.y, origin.z};

    // unsigned int i0, i1, i2, i3;
    uint32_t *pyrs = (uint32_t *)self.pyramids;
    uint64_t i0 = pyrs[ID * 5 + 0];
    uint64_t i1 = pyrs[ID * 5 + 1];
    uint64_t i2 = pyrs[ID * 5 + 2];
    uint64_t i3 = pyrs[ID * 5 + 3];
    uint64_t i4 = pyrs[ID * 5 + 4];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];
    vec3f P4 = self.vertices[i4];

    float S0 = self.scalars[i0];
    float S1 = self.scalars[i1];
    float S2 = self.scalars[i2];
    float S3 = self.scalars[i3];
    float S4 = self.scalars[i4];

    // prd.missed = false;              // for
    // prd.dataValue = S0;              // testing
    // optixReportIntersection(0.f, 0); // please
    // return;                          // remove

    if (interpolatePyramid(P, P0, P1, P2, P3, P4, S0, S1, S2, S3, S4, prd.dataValue))
    {
        optixReportIntersection(0.f, 0);
        prd.missed = false;
        return;
    }
}

OPTIX_INTERSECT_PROGRAM(wedgePointQuery)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    const auto &self = owl::getProgramData<UnstructuredElementData>();
    unsigned int primID = optixGetPrimitiveIndex(); //+ self.offset;
    float3 origin = optixGetObjectRayOrigin();

    // for (int i = 0; i < ELEMENTS_PER_BOX; ++i) {
    //   uint32_t ID = primID * ELEMENTS_PER_BOX + i;
    if (primID >= self.numWedges)
        return;

    // printf("TetrahedraPointQuery: primID = %d\\n", primID);

    unsigned int ID = (uint32_t(primID) /*+ self.offset*/) /* ELEMENTS_PER_BOX*/;

    vec3f P = {origin.x, origin.y, origin.z};

    // unsigned int i0, i1, i2, i3;
    uint32_t *weds = (uint32_t *)self.wedges;
    uint64_t i0 = weds[ID * 6 + 0];
    uint64_t i1 = weds[ID * 6 + 1];
    uint64_t i2 = weds[ID * 6 + 2];
    uint64_t i3 = weds[ID * 6 + 3];
    uint64_t i4 = weds[ID * 6 + 4];
    uint64_t i5 = weds[ID * 6 + 5];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];
    vec3f P4 = self.vertices[i4];
    vec3f P5 = self.vertices[i5];

    float S0 = self.scalars[i0];
    float S1 = self.scalars[i1];
    float S2 = self.scalars[i2];
    float S3 = self.scalars[i3];
    float S4 = self.scalars[i4];
    float S5 = self.scalars[i5];

    // prd.missed = false;              // for
    // prd.dataValue = S0;              // testing
    // optixReportIntersection(0.f, 0); // please
    // return;                          // remove

    if (interpolateWedge(P, P0, P1, P2, P3, P4, P5, S0, S1, S2, S3, S4, S5, prd.dataValue))
    {
        optixReportIntersection(0.f, 0);
        prd.missed = false;
        return;
    }
}

OPTIX_INTERSECT_PROGRAM(hexahedraPointQuery)
()
{
    RayPayload &prd = owl::getPRD<RayPayload>();
    const auto &self = owl::getProgramData<UnstructuredElementData>();
    unsigned int primID = optixGetPrimitiveIndex(); //+ self.offset;
    float3 origin = optixGetObjectRayOrigin();

    // for (int i = 0; i < ELEMENTS_PER_BOX; ++i) {
    //   uint32_t ID = primID * ELEMENTS_PER_BOX + i;
    if (primID >= self.numHexahedra)
        return;

    // printf("TetrahedraPointQuery: primID = %d\\n", primID);

    unsigned int ID = (uint32_t(primID)) /* ELEMENTS_PER_BOX*/;

    vec3f P = {origin.x, origin.y, origin.z};

    // unsigned int i0, i1, i2, i3;
    uint32_t *hexes = (uint32_t *)self.hexahedra;
    uint64_t i0 = hexes[ID * 8 + 0];
    uint64_t i1 = hexes[ID * 8 + 1];
    uint64_t i2 = hexes[ID * 8 + 2];
    uint64_t i3 = hexes[ID * 8 + 3];
    uint64_t i4 = hexes[ID * 8 + 4];
    uint64_t i5 = hexes[ID * 8 + 5];
    uint64_t i6 = hexes[ID * 8 + 6];
    uint64_t i7 = hexes[ID * 8 + 7];

    vec3f P0 = self.vertices[i0];
    vec3f P1 = self.vertices[i1];
    vec3f P2 = self.vertices[i2];
    vec3f P3 = self.vertices[i3];
    vec3f P4 = self.vertices[i4];
    vec3f P5 = self.vertices[i5];
    vec3f P6 = self.vertices[i6];
    vec3f P7 = self.vertices[i7];

    float S0 = self.scalars[i0];
    float S1 = self.scalars[i1];
    float S2 = self.scalars[i2];
    float S3 = self.scalars[i3];
    float S4 = self.scalars[i4];
    float S5 = self.scalars[i5];
    float S6 = self.scalars[i6];
    float S7 = self.scalars[i7];

    // prd.missed = false;              // for
    // prd.dataValue = S0;              // testing
    // optixReportIntersection(0.f, 0); // please
    // return;                          // remove

    if (interpolateHexahedra(P, P0, P1, P2, P3, P4, P5, P6, P7,
                             S0, S1, S2, S3, S4, S5, S6, S7, prd.dataValue))
    {
        optixReportIntersection(0.f, 0);
        prd.missed = false;
        return;
    }
}

  OPTIX_INTERSECT_PROGRAM(volumeIntersection)()
  {
    RayPayload &prd = owl::getPRD<RayPayload>();
    const auto &self = owl::getProgramData<MacrocellData>();
    const int primID = optixGetPrimitiveIndex() + self.offset;

    box4f bbox = self.bboxes[primID];
    float3 lb = make_float3(bbox.lower.x, bbox.lower.y, bbox.lower.z);
    float3 rt = make_float3(bbox.upper.x, bbox.upper.y, bbox.upper.z);
    float3 origin = optixGetObjectRayOrigin();

    // note, this is _not_ normalized. Useful for computing world space tmin/mmax
    float3 direction = optixGetObjectRayDirection();

    // float3 rt = make_float3(mx.x(), mx.y(), mx.z() + 1.f);

    // typical ray AABB intersection test
    float3 dirfrac;

    // direction is unit direction vector of ray
    dirfrac.x = 1.0f / direction.x;
    dirfrac.y = 1.0f / direction.y;
    dirfrac.z = 1.0f / direction.z;

    // lb is the corner of AABB with minimal coordinates - left bottom, rt is maximal corner
    // origin is origin of ray
    float t1 = (lb.x - origin.x)*dirfrac.x;
    float t2 = (rt.x - origin.x)*dirfrac.x;
    float t3 = (lb.y - origin.y)*dirfrac.y;
    float t4 = (rt.y - origin.y)*dirfrac.y;
    float t5 = (lb.z - origin.z)*dirfrac.z;
    float t6 = (rt.z - origin.z)*dirfrac.z;

    float thit0 = max(max(min(t1, t2), min(t3, t4)), min(t5, t6));
    float thit1 = min(min(max(t1, t2), max(t3, t4)), max(t5, t6));

    // if tmax < 0, ray (line) is intersecting AABB, but the whole AABB is behind us
    if (thit1 < 0) { return; }

    // if tmin > tmax, ray doesn't intersect AABB
    if (thit0 >= thit1) { return; }

    // clip hit to near position
    thit0 = max(thit0, optixGetRayTmin());

    if (optixReportIntersection(thit0, /* hit kind */ 0)) 
    {
      prd.t0 = max(prd.t0, thit0);
      prd.t1 = min(prd.t1, thit1);
    }
  }



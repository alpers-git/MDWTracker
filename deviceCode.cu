#include "hip/hip_runtime.h"
#include "deviceCode.h"
#include <optix_device.h>
#include "renderer.h"

using namespace owl;
using namespace dtracker;

extern "C" __constant__ static LaunchParams optixLaunchParams;

#define DEBUG 0
// create a debug function macro that gets called only for center pixel
inline __device__ bool dbg()
{
  auto lp = optixLaunchParams;
#if DEBUG
  return false;
#else
  auto pixelID = vec2i(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
  return (lp.fbSize.x / 2 == pixelID.x) &&
         (lp.fbSize.y / 2 == pixelID.y);
#define ACTIVATE_CROSSHAIRS
#endif
}

inline __device__ void generateRay(const vec2f screen, owl::Ray &ray)
{
    auto &lp = optixLaunchParams;
    ray.origin = lp.camera.origin;
    vec3f direction = lp.camera.lower_left_corner +
                      screen.u * lp.camera.horizontal +
                      screen.v * lp.camera.vertical;
    // direction = normalize(direction);
    if (fabs(direction.x) < 1e-5f)
        direction.x = 1e-5f;
    if (fabs(direction.y) < 1e-5f)
        direction.y = 1e-5f;
    if (fabs(direction.z) < 1e-5f)
        direction.z = 1e-5f;
    ray.direction = normalize(direction - ray.origin);
}

inline __device__
vec3f missColor(const Ray &ray)
{
    const vec2i pixelID = owl::getLaunchIndex();
    
    // Calculate the intersection point in world coordinates
    vec3f intersectionPoint = ray.origin + ray.direction * 1e20f;
    
    // Calculate the grid pattern based on the intersection point
    int pattern = ((int)intersectionPoint.x / 18) ^ ((int)intersectionPoint.y / 18);
    
    vec3f color = (pattern & 1) ? vec3f(.2f, .2f, .26f) : vec3f(.1f, .1f, .16f);
    return color;
}

// Simple raygen that creates a checker-board pattern
OPTIX_RAYGEN_PROGRAM(testRayGen)
()
{
    auto &lp = optixLaunchParams;
    const vec2i pixelID = owl::getLaunchIndex();
    const int fbOfs = pixelID.x + lp.fbSize.x * pixelID.y;

    const vec2f screen = (vec2f(pixelID) + vec2f(0.5f)) / vec2f(lp.fbSize);
    Ray ray;
    generateRay(screen, ray);

    RayPayload prd;

    traceRay(/*accel to trace against*/ lp.volume.elementTLAS,
             /*the ray to trace*/ ray,
             /*prd*/ prd);
             
    //prd.rgba = vec4f(missColor(ray), 1);
    // Choose the appropriate color based on the checkerboard pattern
    lp.fbPtr[fbOfs] = owl::make_rgba(prd.rgba);
}

OPTIX_MISS_PROGRAM(miss)
()
{
    const vec2i pixelID = owl::getLaunchIndex();

    const MissProgData &self = owl::getProgramData<MissProgData>();

    RayPayload &prd = owl::getPRD<RayPayload>();
    int pattern = (pixelID.x / 18) ^ (pixelID.y / 18);
    prd.rgba = (pattern & 1) ? vec4f(self.color1, 1) : vec4f(self.color0, 1);
    prd.missed = true;
}